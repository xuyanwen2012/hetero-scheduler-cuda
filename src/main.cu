#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <Common/helper_cuda.hpp>  // helper functions for CUDA error checking and initialization
#include <algorithm>
#include <functional>
#include <iostream>
#include <numeric>

#include "omp.h"

__global__ void EmptyKernel() {}

// __global__ void do_some_work(float* in, float* out, const std::size_t n) {
//   const auto i = blockIdx.x * blockDim.x + threadIdx.x;
//   if (i < n) {
//     float temp = in[i];
//     for (int j = 0; j < 1000; ++j) {
//       temp = cos(sin(temp)) * tan(temp);  // Expensive trigonometric
//       operations
//     }
//     out[i] = temp;
//   }
// }

__device__ float do_some_work_func(float temp) {
  for (int j = 0; j < 1000; ++j) {
    temp = cos(sin(temp)) * tan(temp);  // Expensive trigonometric operations
  }
  return temp;
}

__global__ void kernel1(const float* in, float* out, const int which_sm) {
  const auto i = threadIdx.x;
  out[i] = do_some_work_func(in[i]) + which_sm * 1000000;
}

void MeasureCudaKernel(const std::function<void()>& kernel_func,
                       const char* kernel_name = "CUDA Kernel") {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);

  kernel_func();

  hipEventRecord(stop, nullptr);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << kernel_name << " - Elapsed time: " << milliseconds << " ms\n";

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

int main() {
  constexpr std::size_t n = 10'000'000;

  // Duck has 8 SMs
  // Toucan has 36 SMs
  constexpr auto num_sm = 8;

  float* data;
  float* out_data;
  float* out_data_2;

  checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&out_data, n * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&out_data_2, n * sizeof(float)));

  constexpr auto threads = 4;
  constexpr auto num_blocks = 1;
  constexpr auto num_threads_per_block = 128;

  auto streams = new hipStream_t[threads];
  for (int i = 0; i < threads; i++) {
    checkCudaErrors(hipStreamCreate(&streams[i]));
  }

  // initialize data
  std::iota(data, data + n, 0.0f);

  // constexpr std::size_t numThreadsPerBlock = 256;
  // constexpr std::size_t numBlocks = 16;  // blocks per SM

  // warmup
  EmptyKernel<<<num_blocks, num_threads_per_block>>>();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);

  // measureCudaKernel([&]() {
#pragma omp parallel for
  for (int i = 0; i < threads; ++i) {
    printf("Thread %d\n", i);
    const auto offset = i * 128;
    kernel1<<<1, 128, 0, streams[i]>>>(data + offset, out_data + offset, i);
  }

  hipEventRecord(stop, nullptr);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << " - Elapsed time: " << milliseconds << " ms\n";

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // });

  // -----------------------------------

  // Wait for the kernel to finish
  // checkCudaErrors(hipDeviceSynchronize());

  // peek 10 results
  std::cout << "Results: ";
  std::for_each_n(out_data, 10, [](const auto& x) { std::cout << x << "\n"; });

  checkCudaErrors(hipFree(data));
  checkCudaErrors(hipFree(out_data));
  checkCudaErrors(hipFree(out_data_2));

  for (int i = 0; i < threads + 1; i++) {
    hipStreamDestroy(streams[i]);
  }

  delete[] streams;

  return 0;
}