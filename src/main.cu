#include "hip/hip_runtime.h"
#include <CLI/CLI.hpp>
#include <Common/helper_cuda.hpp>  // helper functions for CUDA error checking and initialization
#include <algorithm>
#include <functional>
#include <iostream>
#include <numeric>

#include "omp.h"

__global__ void emptyKernel() {}

// __global__ void do_some_work(float* in, float* out, const std::size_t n) {
//   const auto i = blockIdx.x * blockDim.x + threadIdx.x;
//   if (i < n) {
//     float temp = in[i];
//     for (int j = 0; j < 1000; ++j) {
//       temp = cos(sin(temp)) * tan(temp);  // Expensive trigonometric
//       operations
//     }
//     out[i] = temp;
//   }
// }

__device__ float do_some_work_func(float temp) {
  for (int j = 0; j < 1000; ++j) {
    temp = cos(sin(temp)) * tan(temp);  // Expensive trigonometric operations
  }
  return temp;
}

__global__ void kernel1(float* in, float* out, int which_sm) {
  const auto i = threadIdx.x;
  out[i] = do_some_work_func(in[i]) + which_sm * 1000000;
}

void measureCudaKernel(std::function<void()> kernelFunc,
                       const char* kernelName = "CUDA Kernel") {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);

  kernelFunc();

  hipEventRecord(stop, nullptr);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << kernelName << " - Elapsed time: " << milliseconds << " ms\n";

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

int main(int argc, char** argv) {
  CLI::App app("My C++ App");

  int num_threads = 1;

  app.add_option("-t,--threads", num_threads, "Number of threads")
      ->default_val(1);

  CLI11_PARSE(app, argc, argv);

  omp_set_num_threads(num_threads);

  constexpr std::size_t n = 10'000'000;

  // Duck has 8 SMs
  // Toucan has 36 SMs
  // constexpr auto num_sm = 8;

  float* data;
  float* out_data;
  float* out_data_2;

  checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&out_data, n * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&out_data_2, n * sizeof(float)));

  hipStream_t* streams = new hipStream_t[num_threads];
  for (int i = 0; i < num_threads; i++) {
    checkCudaErrors(hipStreamCreate(&streams[i]));
  }

  // initialize data
  std::iota(data, data + n, 0.0f);

  // constexpr std::size_t numThreadsPerBlock = 256;
  // constexpr std::size_t numBlocks = 16;  // blocks per SM

  // warmup
  emptyKernel<<<1, 1>>>();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);

#pragma omp parallel for 
  for (int i = 0; i < num_threads; ++i) {
    printf("Thread %d\n", i);
    const auto offset = i * 128;
    kernel1<<<1, 128, 0, streams[i]>>>(data + offset, out_data + offset, i);
  }

  hipEventRecord(stop, nullptr);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << " - Elapsed time: " << milliseconds << " ms\n";

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // });

  // -----------------------------------

  // Wait for the kernel to finish
  // checkCudaErrors(hipDeviceSynchronize());

  // peek 10 results
  std::cout << "Results: ";
  std::for_each_n(out_data, 10, [](const auto& x) { std::cout << x << "\n"; });

  checkCudaErrors(hipFree(data));
  checkCudaErrors(hipFree(out_data));
  checkCudaErrors(hipFree(out_data_2));

  for (int i = 0; i < num_threads + 1; i++) {
    hipStreamDestroy(streams[i]);
  }

  delete[] streams;

  return 0;
}