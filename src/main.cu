#include "hip/hip_runtime.h"
#include <Common/helper_cuda.hpp>  // helper functions for CUDA error checking and initialization
#include <algorithm>
#include <iostream>
#include <numeric>

__global__ void emptyKernel() {}

__global__ void do_some_work(float* in, float* out, const std::size_t n) {
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float temp = in[i];
    for (int j = 0; j < 1000; ++j) {
      temp = cos(sin(temp)) * tan(temp);  // Expensive trigonometric operations
    }
    out[i] = temp;
  }
}

__global__ void myKernel() {
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  printf("Thread %d is running on SM %d\n", threadId, blockIdx.x / 4);
}

int main() {
  constexpr std::size_t n = 10'000'000;

  float* data;
  float* out_data;

  checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&out_data, n * sizeof(float)));

  // initialize data
  std::iota(data, data + n, 0.0f);

  constexpr std::size_t numThreadsPerBlock = 256;
  constexpr std::size_t numBlocks = 16;  // blocks per SM

  // warmup
  emptyKernel<<<numBlocks, numThreadsPerBlock>>>();

  // -----------------------------------
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);

  do_some_work<<<numBlocks, numThreadsPerBlock>>>(data, out_data, n);

  hipEventRecord(stop, nullptr);

  hipEventSynchronize(stop);

  // Calculate the elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time: " << milliseconds << " ms\n";

  // -----------------------------------

  // Wait for the kernel to finish
  checkCudaErrors(hipDeviceSynchronize());

  // peek 10 results
  std::cout << "Results: ";
  std::for_each_n(out_data, 10, [](const auto& x) { std::cout << x << "\n"; });

  hipEventDestroy(start);
  hipEventDestroy(stop);

  checkCudaErrors(hipFree(data));
  checkCudaErrors(hipFree(out_data));

  return 0;
}